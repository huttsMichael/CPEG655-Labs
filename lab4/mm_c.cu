
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// CUDA kernel for matrix multiplication using tiling algorithm
__global__ void matrixMulTiled(float *C, float *A, float *B, int N, int tile_size) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // identify the starting point of the current tile in the input matrices
    int row = by * tile_size + ty;
    int col = bx * tile_size + tx;

    // dynamic shared memory allocation for storing a tile of matrix A and B
    extern __shared__ float sharedMemory[];
    float *As = (float*)&sharedMemory[0];
    float *Bs = (float*)&sharedMemory[tile_size * tile_size];

    float Cvalue = 0.0f;

    // loop over tiles of input matrices
    for (int t = 0; t < N / tile_size; ++t) {
        // load the tiles of matrices A and B into shared memory
        As[ty * tile_size + tx] = A[row * N + t * tile_size + tx];
        Bs[ty * tile_size + tx] = B[(t * tile_size + ty) * N + col];

        // synchronize to make sure the tiles are loaded
        __syncthreads();

        // multiply the tiles and accumulate the result
        for (int k = 0; k < tile_size; ++k)
            Cvalue += As[ty * tile_size + k] * Bs[k * tile_size + tx];

        // synchronize before loading the next tiles
        __syncthreads();
    }

    // write the result to the output matrix
    C[row * N + col] = Cvalue;
}

// host code for matrix multiplication
void mm(float *C, float *A, float *B, int N) {
    for (int j = 0; j < N; j++)
        for (int i = 0; i < N; i++)
            for (int k = 0; k < N; k++)
                C[i * N + j] += A[i * N + k] * B[k * N + j];
}

// function to calculate root mean square error 
float calculateRMSE(float *A, float *B, int size) {
    float sumSquaredDiff = 0.0f;

    // iterate through all elements of the matrices
    for (int i = 0; i < size; i++) {
        // calculate the difference between corresponding elements of matrices A and B
        float diff = A[i] - B[i];

        sumSquaredDiff += diff * diff;
    }

    // calculate the mean squared difference
    float meanSquaredDiff = sumSquaredDiff / size;

    // calculate the square root of the mean squared difference to get RMSE
    float rmse = sqrtf(meanSquaredDiff);

    return rmse;
}

int main(int argc, char **argv) {
    struct timeval begin, end;
    int NB_values[] = {2, 4, 8, 16, 32};
    int NT_values[] = {2, 4, 8, 16, 32};
    int num_runs = 10; 

    // run for all inputs multiple times times over
    for (int nbIndex = 0; nbIndex < 5; nbIndex++) {
        for (int ntIndex = 0; ntIndex < 5; ntIndex++) {
            int NB = NB_values[nbIndex];
            int NT = NT_values[ntIndex];

            // calculate NK based on the relationship N^2 = NB^2 * NT^2 * NK^2
            int NK = sqrt((2048 * 2048) / (NB * NB * NT * NT));

            int N = 2048;
            int size = N * N;

            double total_gpu_time = 0.0;

            for (int run = 0; run < num_runs; run++) {
                // allocate host memory
                float *h_A = (float *)malloc(size * sizeof(float));
                float *h_B = (float *)malloc(size * sizeof(float));
                float *h_C_gpu = (float *)malloc(size * sizeof(float));

                // allocate device memory
                float *d_A, *d_B, *d_C;
                hipMalloc((void **)&d_A, size * sizeof(float));
                hipMalloc((void **)&d_B, size * sizeof(float));
                hipMalloc((void **)&d_C, size * sizeof(float));

                // copy host matrices to device
                hipMemcpy(d_A, h_A, size * sizeof(float), hipMemcpyHostToDevice);
                hipMemcpy(d_B, h_B, size * sizeof(float), hipMemcpyHostToDevice);

                // set up the execution configuration
                dim3 threadsPerBlock(NT, NT);
                dim3 numBlocks(NK, NK);

                // measure the computation time for GPU version
                gettimeofday(&begin, NULL);

                // launch the CUDA kernel
                matrixMulTiled<<<numBlocks, threadsPerBlock, 2 * NB * NT * sizeof(float)>>>(d_C, d_A, d_B, N, NB);

                // wait for the kernel to finish
                hipDeviceSynchronize();

                gettimeofday(&end, NULL);

                double elapsed_time = (end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec) * 1.0 / 1000000;
                total_gpu_time += elapsed_time;

                // fprintf(stdout, "run %d - GPU time for N=%d, NB=%d, NT=%d: %lf\n", run + 1, N, NB, NT, elapsed_time);

                // copy the result back to the host
                hipMemcpy(h_C_gpu, d_C, size * sizeof(float), hipMemcpyDeviceToHost);

                // free device and host memory
                hipFree(d_A);
                hipFree(d_B);
                hipFree(d_C);
                free(h_A);
                free(h_B);
                free(h_C_gpu);
            }

            double avg_gpu_time = total_gpu_time / num_runs;
            fprintf(stdout, "Average GPU time for N=%d, NB=%d, NT=%d: %lf\n", N, NB, NT, avg_gpu_time);
        }
    }

    return 0;
}
