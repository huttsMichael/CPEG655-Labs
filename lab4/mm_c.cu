
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// CUDA kernel for matrix multiplication using tiling algorithm
__global__ void matrixMulTiled(float *C, float *A, float *B, int N, int tile_size) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Identify the starting point of the current tile in the input matrices
    int row = by * tile_size + ty;
    int col = bx * tile_size + tx;

    // Dynamic shared memory allocation for storing a tile of matrix A and B
    extern __shared__ float sharedMemory[];
    float *As = (float*)&sharedMemory[0];
    float *Bs = (float*)&sharedMemory[tile_size * tile_size];

    float Cvalue = 0.0f;

    // Loop over tiles of input matrices
    for (int t = 0; t < N / tile_size; ++t) {
        // Load the tiles of matrices A and B into shared memory
        As[ty * tile_size + tx] = A[row * N + t * tile_size + tx];
        Bs[ty * tile_size + tx] = B[(t * tile_size + ty) * N + col];

        // Synchronize to make sure the tiles are loaded
        __syncthreads();

        // Multiply the tiles and accumulate the result
        for (int k = 0; k < tile_size; ++k)
            Cvalue += As[ty * tile_size + k] * Bs[k * tile_size + tx];

        // Synchronize before loading the next tiles
        __syncthreads();
    }

    // Write the result to the output matrix
    C[row * N + col] = Cvalue;
}


// Host code for matrix multiplication
void mm(float *C, float *A, float *B, int N) {
    for (int j = 0; j < N; j++)
        for (int i = 0; i < N; i++)
            for (int k = 0; k < N; k++)
                C[i * N + j] += A[i * N + k] * B[k * N + j];
}

// Function to calculate Root Mean Square Error (RMSE) between two matrices
float calculateRMSE(float *A, float *B, int size) {
    float sumSquaredDiff = 0.0f;
    for (int i = 0; i < size; i++) {
        float diff = A[i] - B[i];
        sumSquaredDiff += diff * diff;
    }
    float meanSquaredDiff = sumSquaredDiff / size;
    return sqrtf(meanSquaredDiff);
}

int main(int argc, char **argv) {
    struct timeval begin, end;
    int NB_values[] = {8, 16, 32};
    int NT_values[] = {8, 16, 32};
    
    for (int nbIndex = 0; nbIndex < 3; nbIndex++) {
        for (int ntIndex = 0; ntIndex < 3; ntIndex++) {
            int NB = NB_values[nbIndex];
            int NT = NT_values[ntIndex];

            // Calculate NK based on the relationship N^2 = NB^2 * NT^2 * NK^2
            int NK = sqrt(2048 * 2048 / (NB * NB * NT * NT));

            int N = 2048;
            int size = N * N;

            // Allocate host memory
            float *h_A = (float *)malloc(size * sizeof(float));
            float *h_B = (float *)malloc(size * sizeof(float));
            float *h_C_gpu = (float *)malloc(size * sizeof(float));

            // Allocate device memory
            float *d_A, *d_B, *d_C;
            hipMalloc((void **)&d_A, size * sizeof(float));
            hipMalloc((void **)&d_B, size * sizeof(float));
            hipMalloc((void **)&d_C, size * sizeof(float));

            // Copy host matrices to device
            hipMemcpy(d_A, h_A, size * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(d_B, h_B, size * sizeof(float), hipMemcpyHostToDevice);

            // Set up the execution configuration
            dim3 threadsPerBlock(NT, NT);
            dim3 numBlocks(NK, NK);

            // Measure the computation time for GPU version
            gettimeofday(&begin, NULL);

            // Launch the CUDA kernel
            matrixMulTiled<<<numBlocks, threadsPerBlock>>>(d_C, d_A, d_B, N, NB);

            // Wait for the kernel to finish
            hipDeviceSynchronize();

            gettimeofday(&end, NULL);

            fprintf(stdout, "GPU time for N=%d, NB=%d, NT=%d: %lf\n", N, NB, NT, (end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec) * 1.0 / 1000000);

            // Copy the result back to the host
            hipMemcpy(h_C_gpu, d_C, size * sizeof(float), hipMemcpyDeviceToHost);

            // Free device and host memory
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_C);
            free(h_A);
            free(h_B);
            free(h_C_gpu);
        }
    }

    return 0;
}