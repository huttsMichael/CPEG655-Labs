
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

// CUDA kernel for matrix multiplication
__global__ void matrixMul(float *C, float *A, float *B, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Initialize a variable to store the sum for the current element of matrix C
    float sum = 0.0f;

    // Perform the actual matrix multiplication for the current element (i, j)
    for (int k = 0; k < N; ++k) {
        // Multiply corresponding elements from matrices A and B and accumulate the result
        sum += A[i * N + k] * B[k * N + j];
    }

    // Store the final result in the corresponding element of matrix C
    C[i * N + j] = sum;
}

// Host code for matrix multiplication
void mm(float *C, float *A, float *B, int N) {
    for (int j = 0; j < N; j++)
        for (int i = 0; i < N; i++)
            for (int k = 0; k < N; k++)
                C[i * N + j] += A[i * N + k] * B[k * N + j];
}

// Function to calculate Root Mean Square Error (RMSE) between two matrices
float calculateRMSE(float *A, float *B, int size) {
    // Initialize variable to store the sum of squared differences
    float sumSquaredDiff = 0.0f;

    // Iterate through all elements of the matrices
    for (int i = 0; i < size; i++) {
        // Calculate the difference between corresponding elements of matrices A and B
        float diff = A[i] - B[i];

        // Accumulate the squared difference
        sumSquaredDiff += diff * diff;
    }

    // Calculate the mean squared difference
    float meanSquaredDiff = sumSquaredDiff / size;

    // Calculate the square root of the mean squared difference to get RMSE
    float rmse = sqrtf(meanSquaredDiff);

    // Return the calculated RMSE
    return rmse;
}


int main(int argc, char **argv) {
    struct timeval begin, end;
    int sizes[] = {16, 32};

    // Run for both sizes
    for (int sizeIndex = 0; sizeIndex < 2; sizeIndex++) {
        int N = sizes[sizeIndex];
        int size = N * N;

        // Allocate host memory
        float *h_A = (float *)malloc(size * sizeof(float));
        float *h_B = (float *)malloc(size * sizeof(float));
        float *h_C_cpu = (float *)malloc(size * sizeof(float));
        float *h_C_gpu = (float *)malloc(size * sizeof(float));

        // Allocate device memory
        float *d_A, *d_B, *d_C;
        hipMalloc((void **)&d_A, size * sizeof(float));
        hipMalloc((void **)&d_B, size * sizeof(float));
        hipMalloc((void **)&d_C, size * sizeof(float));

        // Copy host matrices to device
        hipMemcpy(d_A, h_A, size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size * sizeof(float), hipMemcpyHostToDevice);

        // Set up the execution configuration
        dim3 threadsPerBlock(32, 32); // max threads per block = 1024, 32*32=1024
        dim3 numBlocks(1, 1); // Use only one thread block

        // Measure the computation time for GPU version
        gettimeofday(&begin, NULL);

        // Launch the CUDA kernel
        matrixMul<<<numBlocks, threadsPerBlock>>>(d_C, d_A, d_B, N);


        // Measure the computation time for GPU version
        gettimeofday(&begin, NULL);

        // Launch the CUDA kernel
        matrixMul<<<numBlocks, threadsPerBlock>>>(d_C, d_A, d_B, N);

        // Wait for the kernel to finish
        hipDeviceSynchronize();

        gettimeofday(&end, NULL);

        fprintf(stdout, "GPU time for N=%d: %lf\n", N, (end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec) * 1.0 / 1000000);

        // Copy the result back to the host
        hipMemcpy(h_C_gpu, d_C, size * sizeof(float), hipMemcpyDeviceToHost);

        // Measure the computation time for CPU version
        gettimeofday(&begin, NULL);

        // Call the CPU matrix multiplication function
        mm(h_C_cpu, h_A, h_B, N);

        gettimeofday(&end, NULL);

        fprintf(stdout, "CPU time for N=%d: %lf\n", N, (end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec) * 1.0 / 1000000);

        // Verify the correctness by calculating RMSE
        float rmse = calculateRMSE(h_C_cpu, h_C_gpu, size);
        fprintf(stdout, "RMSE for N=%d: %e\n", N, rmse);

        // Free device and host memory
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        free(h_A);
        free(h_B);
        free(h_C_cpu);
        free(h_C_gpu);
    }

    return 0;
}
