
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

// CUDA kernel for matrix multiplication
__global__ void matrixMul(float *C, float *A, float *B, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; ++k) {
            sum += A[i * N + k] * B[k * N + j];
        }
        C[i * N + j] = sum;
    }
}

// Host code for matrix multiplication
void mm(float *C, float *A, float *B, int N) {
    for (int j = 0; j < N; j++)
        for (int i = 0; i < N; i++)
            for (int k = 0; k < N; k++)
                C[i * N + j] += A[i * N + k] * B[k * N + j];
}

// Function to calculate Root Mean Square Error (RMSE)
float calculateRMSE(float *A, float *B, int size) {
    float sumSquaredDiff = 0.0f;
    for (int i = 0; i < size; i++) {
        float diff = A[i] - B[i];
        sumSquaredDiff += diff * diff;
    }
    float meanSquaredDiff = sumSquaredDiff / size;
    return sqrtf(meanSquaredDiff);
}

int main(int argc, char **argv) {
    struct timeval begin, end;
    int N = 256;
    int size = N * N;

    // Allocate host memory
    float *h_A = (float *)malloc(size * sizeof(float));
    float *h_B = (float *)malloc(size * sizeof(float));
    float *h_C_cpu = (float *)malloc(size * sizeof(float));
    float *h_C_gpu = (float *)malloc(size * sizeof(float));

    // Initialize matrices A and B
    // ... (you can replace this with your own initialization)

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size * sizeof(float));
    hipMalloc((void **)&d_B, size * sizeof(float));
    hipMalloc((void **)&d_C, size * sizeof(float));

    // Copy host matrices to device
    hipMemcpy(d_A, h_A, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size * sizeof(float), hipMemcpyHostToDevice);

    // Set up the execution configuration
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

    // Measure the computation time for GPU version
    gettimeofday(&begin, NULL);

    // Launch the CUDA kernel
    matrixMul<<<numBlocks, threadsPerBlock>>>(d_C, d_A, d_B, N);

    // Wait for the kernel to finish
    hipDeviceSynchronize();

    gettimeofday(&end, NULL);

    fprintf(stdout, "GPU time = %lf\n", (end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec) * 1.0 / 1000000);

    // Copy the result back to the host
    hipMemcpy(h_C_gpu, d_C, size * sizeof(float), hipMemcpyDeviceToHost);

    // Measure the computation time for CPU version
    gettimeofday(&begin, NULL);

    // Call the CPU matrix multiplication function
    mm(h_C_cpu, h_A, h_B, N);

    gettimeofday(&end, NULL);

    fprintf(stdout, "CPU time = %lf\n", (end.tv_sec - begin.tv_sec) + (end.tv_usec - begin.tv_usec) * 1.0 / 1000000);

    // Verify the correctness by calculating RMSE
    float rmse = calculateRMSE(h_C_cpu, h_C_gpu, size);
    fprintf(stdout, "RMSE = %e\n", rmse);

    // Free device and host memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C_cpu);
    free(h_C_gpu);

    return 0;
}
